
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add_vec_cuda(int *a, int *b, int *c){

    int index = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (index < 100){
        c[index] = a[index] + b[index]; 
    }


}

int* add_vec(int *a, int *b){

    int *d_a, *d_b, *d_c;
    int *c;

    c = (int*) malloc(sizeof(int) * 100);

    int size = 100 * sizeof(int);

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);

    add_vec_cuda<<<1, 100>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);


    return c;
}