
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void from_gpu(void){

    printf("%d, %d\n", blockIdx.x, threadIdx.x);

}

int main(){

    from_gpu<<<10, 10>>>();
    hipDeviceSynchronize();

    return 0;
}