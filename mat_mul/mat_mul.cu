
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void mul_mat(int *a, int *b, int *c, int n, int m){

    // a = matrix A
    // b = matrix B
    // c = matrix C

    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    int value = 0;

    if (x > n || y > m){
        return;
    }

    for(int i = 0; i < 1024; i++){
        value += a[x * n + i] * b[y + m * i];
    }

    c[x * n + y] = value;

}

void mat_mul_cuda(int *a, int *b, int *c, int n, int m){

    int *d_a, *d_b, *d_c;

    size_t size = m * n * sizeof(int);
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);

    int threads = 16;
    int blocks = (n + threads - 1) / threads;
    dim3 BLOCKS (blocks, blocks);
    dim3 THREADS(threads, threads);
    mul_mat<<<BLOCKS, THREADS>>>(d_a, d_b, d_c, n, m);
    hipDeviceSynchronize();

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

}